#include "hip/hip_runtime.h"
#include <array>
#include <Random123/boxmuller.hpp>

#include <arbor/arb_types.hpp>
#include <arbor/gpu/gpu_api.hpp>
#include <arbor/gpu/gpu_common.hpp>

#include "backends/rand_impl.hpp"

namespace arb {
namespace gpu {

__global__
void generate_normal_random_values_kernel (
    std::size_t width,
    std::size_t num_variables,
    arb::cbprng::value_type seed, 
    arb::cbprng::value_type mech_id,
    arb::cbprng::value_type counter,
    arb_size_type** prng_indices,
    arb_value_type** dst0,
    arb_value_type** dst1,
    arb_value_type** dst2,
    arb_value_type** dst3
) {
    int const tid = threadIdx.x + blockDim.x*blockIdx.x;
    std::uint64_t const vid = blockIdx.y;

    arb_size_type const* gids = prng_indices[0];
    arb_size_type const* idxs = prng_indices[1];

    if (tid < width) {
        arb::cbprng::value_type const gid = gids[tid];
        arb::cbprng::value_type const idx = idxs[tid];

        const auto r = arb::cbprng::generate_normal_random_values(seed, mech_id, vid, gid, idx, counter);

        dst0[vid][tid] = r[0];
        dst1[vid][tid] = r[1];
        dst2[vid][tid] = r[2];
        dst3[vid][tid] = r[3];
    }
}

void generate_normal_random_values(
    std::size_t width,
    arb::cbprng::value_type seed,
    arb::cbprng::value_type mech_id,
    arb::cbprng::value_type counter,
    memory::device_vector<arb_size_type*>& prng_indices,
    std::array<memory::device_vector<arb_value_type*>, arb::prng_cache_size()>& dst
)
{
    unsigned const block_dim = 128;
    unsigned const grid_dim_x = impl::block_count(width, block_dim);
    unsigned const grid_dim_y = num_variables;

    generate_normal_random_values_kernel<<<dim3{grid_dim_x, grid_dim_y, 1}, block_dim>>>(
        width,
        dst[0].size(),
        seed, 
        mech_id,
        counter,
        prng_indices.data(),
        dst[0].data(), dst[1].data(), dst[2].data(), dst[3].data()
    );
}

} // namespace gpu
} // namespace arb
