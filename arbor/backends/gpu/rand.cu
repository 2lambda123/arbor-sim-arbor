#include "hip/hip_runtime.h"
#include <array>
#include <Random123/boxmuller.hpp>
#include <Random123/threefry.h>

#include <arbor/arb_types.hpp>
#include <arbor/gpu/gpu_api.hpp>
#include <arbor/gpu/gpu_common.hpp>

namespace arb {
namespace gpu {

__global__
void generate_normal_random_values_kernel (
    std::size_t   width,
    std::size_t   num_variables,
    std::uint64_t seed, 
    std::uint64_t mech_id,
    std::uint64_t counter,
    arb_size_type** prng_indices,
    arb_value_type** dst0,
    arb_value_type** dst1,
    arb_value_type** dst2,
    arb_value_type** dst3
) {

    int const tid = threadIdx.x + blockDim.x*blockIdx.x;
    std::uint64_t const vid = blockIdx.y;

    arb_size_type const* gids = prng_indices[0];
    arb_size_type const* idxs = prng_indices[1];

    if (tid < width) {
        std::uint64_t const gid = gids[tid];
        std::uint64_t const idx = idxs[tid];
        using rng = r123::Threefry4x64_R<12>;
        using counter_type = typename rng::key_type;
        using key_type = typename rng::key_type;

        static_assert(std::is_same<typename counter_type::value_type, std::uint64_t>::value,
            "64 bit width");
        static_assert(std::is_same<typename key_type::value_type, std::uint64_t>::value,
            "64 bit width");
        static_assert(counter_type::static_size == 4, "size of array");
        static_assert(key_type::static_size == 4, "size of array");

        counter_type c{seed, mech_id, vid, counter};
        key_type k{gid, idx, 0, 0};

        const auto r = rng{}(c, k);
        const auto n0 = r123::boxmuller(r[0], r[1]);
        const auto n1 = r123::boxmuller(r[2], r[3]);

        dst0[vid][tid] = n0.x;
        dst1[vid][tid] = n0.y;
        dst2[vid][tid] = n1.x;
        dst3[vid][tid] = n1.y;
    }
}
 

void generate_normal_random_values(
    std::size_t   width,
    std::size_t   num_variables,
    std::uint64_t seed, 
    std::uint64_t mech_id,
    std::uint64_t counter,
    arb_size_type** prng_indices,
    std::array<arb_value_type**, 4> dst
)
{
    unsigned const block_dim = 128;
    unsigned const grid_dim_x = impl::block_count(width, block_dim);
    unsigned const grid_dim_y = num_variables;

    generate_normal_random_values_kernel<<<dim3{grid_dim_x, grid_dim_y, 1}, block_dim>>>(
    width,
    num_variables,
    seed, 
    mech_id,
    counter,
    prng_indices,
    dst[0], dst[1], dst[2], dst[3]
    );
}

} // namespace gpu
} // namespace arb
