#include "hip/hip_runtime.h"
// GPU kernels and wrappers for shared state methods.

#include <cstdint>

#include <backends/event.hpp>
#include <backends/multi_event_stream_state.hpp>

#include <arbor/gpu/gpu_api.hpp>
#include <arbor/gpu/gpu_common.hpp>

#include "backends/rand_impl.hpp"

namespace arb {
namespace gpu {

namespace kernel {

template <typename T>
__global__ void update_time_to_impl(unsigned n,
                                    T* __restrict__ const time_to,
                                    const T* __restrict__ const time,
                                    T dt,
                                    T tmax) {
    unsigned i = threadIdx.x+blockIdx.x*blockDim.x;
    if (i<n) {
        auto t = time[i]+dt;
        time_to[i] = t<tmax? t: tmax;
    }
}

// Vector/scalar addition: x[i] += v ∀i
template <typename T>
__global__ void add_scalar(unsigned n,
                           T* __restrict__ const x,
                           arb_value_type v) {
    unsigned i = threadIdx.x+blockIdx.x*blockDim.x;
    if (i<n) {
        x[i] += v;
    }
}

template <typename T, typename I>
__global__ void set_dt_impl(      T* __restrict__ dt_intdom,
                            const T* __restrict__ time_to,
                            const T* __restrict__ time,
                            const unsigned ncomp,
                                  T* __restrict__ dt_comp,
                            const I* __restrict__ cv_to_intdom) {
    auto idx = blockIdx.x*blockDim.x + threadIdx.x;
    if (idx < ncomp) {
        const auto ind = cv_to_intdom[idx];
        const auto dt = time_to[ind] - time[ind];
        dt_intdom[ind] = dt;
        dt_comp[idx] = dt;
    }
}

__global__ void take_samples_impl(
    multi_event_stream_state<raw_probe_info> s,
    const arb_value_type* __restrict__ const time,
    arb_value_type* __restrict__ const sample_time,
    arb_value_type* __restrict__ const sample_value)
{
    unsigned i = threadIdx.x+blockIdx.x*blockDim.x;
    if (i<s.n) {
        auto begin = s.ev_data+s.begin_offset[i];
        auto end = s.ev_data+s.end_offset[i];
        for (auto p = begin; p!=end; ++p) {
            sample_time[p->offset] = time[i];
            sample_value[p->offset] = p->handle? *p->handle: 0;
        }
    }
}

__global__
void generate_random_values (
    std::size_t width,
    std::size_t num_variables,
    arb::cbprng::value_type seed, 
    arb::cbprng::value_type mech_id,
    arb::cbprng::value_type counter,
    arb_size_type** prng_indices,
    arb_value_type** dst0,
    arb_value_type** dst1,
    arb_value_type** dst2,
    arb_value_type** dst3
) {
    int const tid = threadIdx.x + blockDim.x*blockIdx.x;
    std::uint64_t const vid = blockIdx.y;

    arb_size_type const* gids = prng_indices[0];
    arb_size_type const* idxs = prng_indices[1];

    if (tid < width) {
        arb::cbprng::value_type const gid = gids[tid];
        arb::cbprng::value_type const idx = idxs[tid];

        const auto r = arb::cbprng::generate_normal_random_values(seed, mech_id, vid, gid, idx, counter);

        dst0[vid][tid] = r[0];
        dst1[vid][tid] = r[1];
        dst2[vid][tid] = r[2];
        dst3[vid][tid] = r[3];
    }
}

} // namespace kernel

using impl::block_count;

void add_scalar(std::size_t n, arb_value_type* data, arb_value_type v) {
    if (!n) return;

    constexpr int block_dim = 128;
    const int nblock = block_count(n, block_dim);
    kernel::add_scalar<<<nblock, block_dim>>>(n, data, v);
}

void update_time_to_impl(
    std::size_t n, arb_value_type* time_to, const arb_value_type* time,
    arb_value_type dt, arb_value_type tmax)
{
    if (!n) return;

    constexpr int block_dim = 128;
    const int nblock = block_count(n, block_dim);
    kernel::update_time_to_impl<<<nblock, block_dim>>>(n, time_to, time, dt, tmax);
}

void set_dt_impl(
    arb_size_type nintdom, arb_size_type ncomp, arb_value_type* dt_intdom, arb_value_type* dt_comp,
    const arb_value_type* time_to, const arb_value_type* time, const arb_index_type* cv_to_intdom)
{
    if (!nintdom || !ncomp) return;

    constexpr int block_dim = 128;
    const int nblock = block_count(ncomp, block_dim);
    kernel::set_dt_impl<<<nblock, block_dim>>>(dt_intdom, time_to, time, ncomp, dt_comp, cv_to_intdom);
}

void take_samples_impl(
    const multi_event_stream_state<raw_probe_info>& s,
    const arb_value_type* time, arb_value_type* sample_time, arb_value_type* sample_value)
{
    if (!s.n_streams()) return;

    constexpr int block_dim = 128;
    const int nblock = block_count(s.n_streams(), block_dim);
    kernel::take_samples_impl<<<nblock, block_dim>>>(s, time, sample_time, sample_value);
}

void generate_normal_random_values(
    std::size_t width,                                        // number of sites
    std::size_t n_vars,                                       // number of variables
    arb::cbprng::value_type seed,                             // simulation seed value
    arb::cbprng::value_type mech_id,                          // mechanism id
    arb::cbprng::value_type counter,                          // step counter
    //memory::device_vector<arb_size_type*>& prng_indices,      // holds the gid and per-cell location indices
    arb_size_type** prng_indices,    // holds the gid and per-cell location indices
    //std::array<memory::device_vector<arb_value_type*>, arb::prng_cache_size()>& dst  // pointers to random number cache
    //arb_value_type** dst0,
    //arb_value_type** dst1,
    //arb_value_type** dst2,
    //arb_value_type** dst3
    std::array<arb_value_type**, cbprng::cache_size()> dst  // pointers to random number cache

)
{
    unsigned const block_dim = 128;
    unsigned const grid_dim_x = block_count(width, block_dim);
    unsigned const grid_dim_y = n_vars; //num_variables;

    kernel::generate_random_values<<<dim3{grid_dim_x, grid_dim_y, 1}, block_dim>>>(
        width,
        //dst[0].size(),
        n_vars,
        seed, 
        mech_id,
        counter,
        //prng_indices.data(),
        prng_indices,
        //dst[0].data(), dst[1].data(), dst[2].data(), dst[3].data()
        //dst0, dst1, dst2, dst3
        dst[0], dst[1], dst[2], dst[3]
    );
}

} // namespace gpu
} // namespace arb
