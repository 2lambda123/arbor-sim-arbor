#include "hip/hip_runtime.h"
#include <numeric>
#include <vector>

#include "gtest.h"

#include <math.hpp>
#include <matrix.hpp>
#include <memory/memory.hpp>
#include <util/span.hpp>

using matrix_type = nest::mc::matrix<nest::mc::gpu::matrix_solver>;
using index_type = matrix_type::size_type;

TEST(matrix, solve_gpu)
{
    using namespace nest::mc;

    using nest::mc::util::make_span;

    // trivial case : 1x1 matrix
    {
        matrix_type m({0});

        memory::fill(m.d(),  2);
        memory::fill(m.u(), -1);
        memory::fill(m.rhs(),1);

        m.solve();

        auto rhs = memory::on_host(m.rhs());

        EXPECT_EQ(rhs[0], 0.5);
    }

    // matrices in the range of 2x2 to 100x100
    {
        using namespace nest::mc;
        for(auto n : make_span(2u,101u)) {
            auto p = std::vector<index_type>(n);
            std::iota(p.begin()+1, p.end(), 0);
            matrix_type m{p};

            EXPECT_EQ(m.size(), n);
            EXPECT_EQ(m.num_cells(), 1u);

            memory::fill(m.d(),  2);
            memory::fill(m.u(), -1);
            memory::fill(m.rhs(),1);

            m.solve();

            auto x = memory::on_host(m.rhs());
            auto err = math::square(std::fabs(2.*x[0] - x[1] - 1.));
            for(auto i : make_span(1,n-1)) {
                err += math::square(std::fabs(2.*x[i] - x[i-1] - x[i+1] - 1.));
            }
            err += math::square(std::fabs(2.*x[n-1] - x[n-2] - 1.));

            EXPECT_NEAR(0., std::sqrt(err), 1e-8);
        }
    }
}
